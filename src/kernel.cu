#include "hip/hip_runtime.h"
#include <vector>
#include <iostream>
#include <hip/hip_runtime.h>
#include <opencv2/opencv.hpp>
#include <opencv2/imgproc.hpp>
#include <opencv2/core/cuda.hpp>
#include <opencv2/cudaimgproc.hpp>
#include "kernel.cuh"
#include <hip/hip_runtime.h>

#define BLOCK_X 16
#define BLOCK_Y 16

__device__ __forceinline__ uchar yuvToGreen(uchar Y, uchar U, uchar V) {
    int c = static_cast<int>(Y) - 16;
    int d = static_cast<int>(U) - 128;
    int e = static_cast<int>(V) - 128;

    int g = (298 * c - 100 * d - 208 * e + 128) >> 8;

    return static_cast<uchar>(min(max(g, 0), 255));
}

__global__ void yuv2green_kernel(const uchar* __restrict__ y,
                                 const uchar* __restrict__ u,
                                 const uchar* __restrict__ v,
                                 uchar* __restrict__ g,
                                 int width,
                                 int height,
                                 size_t pitch_u,
                                 size_t pitch_y,
                                 size_t pitch_g) {
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int x = blockIdx.x * blockDim.x + tx;
    int yIdx = blockIdx.y * blockDim.y + ty;

    if (x >= width || yIdx >= height) return;

    __shared__ uchar shared_u[BLOCK_Y / 2][BLOCK_X / 2];
    __shared__ uchar shared_v[BLOCK_Y / 2][BLOCK_X / 2];
    
    if ((ty % 2 == 0) && (tx % 2 == 0)) {
        int uv_x = blockIdx.x * (BLOCK_X / 2) + tx / 2;
        int uv_y = blockIdx.y * (BLOCK_Y / 2) + ty / 2;

        if (uv_x < width / 2 && uv_y < height / 2) {
            shared_u[ty / 2][tx / 2] = u[uv_y * pitch_u + uv_x];
            shared_v[ty / 2][tx / 2] = v[uv_y * pitch_u + uv_x];
        }
    }
    
    __syncthreads();
    
    uchar Yval = y[yIdx * pitch_y + x];
    
    int sx = tx / 2;
    int sy = ty / 2;
    uchar Uval = shared_u[sy][sx];
    uchar Vval = shared_v[sy][sx];
    
    g[yIdx * pitch_g + x] = yuvToGreen(Yval, Uval, Vval);
}

extern "C"
void launchYUV2Green_CUDA(const uchar* d_y, const uchar* d_u, const uchar* d_v,
                          uchar* d_green, int width, int height,
                          size_t pitch_u, size_t pitch_y, size_t pitch_g,
                          hipStream_t stream)
{
    dim3 threads(BLOCK_X, BLOCK_Y);
    dim3 blocks((width + BLOCK_X - 1) / BLOCK_X,
                (height + BLOCK_Y - 1) / BLOCK_Y);

    yuv2green_kernel<<<blocks, threads, 0, stream>>>(
        d_y, d_u, d_v, d_green,
        width, height,
        pitch_u, pitch_y, pitch_g);
}
